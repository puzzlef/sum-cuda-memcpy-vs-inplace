#include <vector>
#include <cstdio>
#include "src/main.hxx"

using namespace std;




void runSum(int N) {
  int repeat = 5;
  vector<float> x(N);
  for (int i=0; i<N; i++)
    x[i] = 1.0/(i+1);

  // Find Σx using a single thread.
  auto a1 = sumSeq(x, {repeat});
  printf("[%09.3f ms] [%f] sumSeq\n", a1.time, a1.result);

  // Find Σx accelerated using CUDA.
  for (int grid=1024; grid<=GRID_LIMIT; grid*=2) {
    for (int block=32; block<=BLOCK_LIMIT; block*=2) {
      auto a2 = sumCuda(x, {repeat, grid, block});
      printf("[%09.3f ms] [%f] sumCuda<<<%d, %d>>>\n", a2.time, a2.result, grid, block);
    }
  }
}


int main(int argc, char **argv) {
  for (int n=1000000; n<=1000000000; n*=10) {
    printf("# Elements %.0e\n", (double) n);
    runSum(n);
    printf("\n");
  }
  return 0;
}
